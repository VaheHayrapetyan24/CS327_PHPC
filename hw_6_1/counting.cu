
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

#define XDIM 512
#define MUL 4

__host__ __device__ int get_idx(char v) {
    return (v == 'C') + (v == 'G') * 2 + (v == 'T') * 3;
}

__global__ void count(char* d_in, int* d_out, int n) {
    // int idx = threadIdx.x + blockDim.x * blockIdx.x;
    // TODO: Fix if not multiples of 4
    // just don't count the last 3, do it in main

    __shared__ int res[MUL][XDIM];


    // int cur_res[4];

    int tid = threadIdx.x;

    char* cur_d_in = d_in + XDIM * blockIdx.x * MUL;


    res[get_idx(cur_d_in[tid * MUL])][tid]++;
    res[get_idx(cur_d_in[tid * MUL + 1])][tid]++;
    res[get_idx(cur_d_in[tid * MUL + 2])][tid]++;
    res[get_idx(cur_d_in[tid * MUL + 3])][tid]++;
    // you'll need to go to 8 if you want to try

    for (int stride = XDIM / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            res[0][tid] += res[0][tid + stride];
            res[1][tid] += res[1][tid + stride];
            res[2][tid] += res[2][tid + stride];
            res[3][tid] += res[3][tid + stride];
        }

        __syncthreads();
    }


    if (tid == 0) {
        d_out[blockIdx.x * 4] = res[0][0];
        d_out[blockIdx.x * 4 + 1] = res[1][0];
        d_out[blockIdx.x * 4 + 2] = res[2][0];
        d_out[blockIdx.x * 4 + 3] = res[3][0];
    }
}

double cpuSecond() {
    struct timeval tp;
    gettimeofday(&tp,NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}

int main() {
    int N = 1 << 23;
    size_t in_size = sizeof(char) * N;
    char *h_in, *d_in;
    int *h_res, *d_res;

    dim3 grid, block;
    block.x = XDIM;
    grid.x = (N + block.x - 1) / (4 * block.x);

    size_t out_size = sizeof(int) * grid.x * 4;
    
    hipHostMalloc(&h_in, in_size, hipHostMallocDefault);
    h_res = (int*) malloc(out_size);

    hipMalloc(&d_in, in_size);
    hipMalloc(&d_res, out_size);

    int r;
    for (int i = 0; i < N; ++i) {
        r = rand() % 4;
        switch (r)
        {
        case 0:
            h_in[i] = 'A';
            break;
        case 1:
            h_in[i] = 'C';
            break;
        case 2:
            h_in[i] = 'G';
            break;
        case 3:
            h_in[i] = 'T';
            break;
        default:
            break;
        }
    }

    double i_start, i_time;
    
    // STARTS HERE
    i_start = cpuSecond();
    hipMemcpy(d_in, h_in, in_size, hipMemcpyHostToDevice);

    count<<<grid, block>>>(d_in, d_res, N);

    hipDeviceSynchronize();

    hipMemcpy(h_res, d_res, out_size, hipMemcpyDeviceToHost);
    
    int device_sum[4] = {0,0,0,0};
    for (int i = 0; i < grid.x * 4; ++i) {
        device_sum[i % 4] += h_res[i];
    }

    i_time = cpuSecond() - i_start;

    printf("device took %f s\n", i_time);

    // ENDS HERE

    

    i_start = cpuSecond();
    int actual_res[4] = {0,0,0,0};
    for (int i = 0; i < N; ++i) {
        actual_res[get_idx(h_in[i])]++;
    }
    i_time = cpuSecond() - i_start;

    printf("host took %f s\n", i_time);




    // printf("the string: ");
    // for (int i = 0; i < N; ++i) {
    //     printf("%c ", h_in[i]);
    // }
    printf("\n");

    printf("host results: ");
    for (int i = 0; i < 4; ++i) {
        printf("%d ", actual_res[i]);
    }
    printf("\n");

    printf("device results: ");
    for (int i = 0; i < 4; ++i) {
        printf("%d ", device_sum[i]);
    }
    printf("\n");

    // for (int i = 0; i < grid.x * 4; ++i) {
    //     printf("%d ", h_res[i]);
    // }
    // printf("\n");













}