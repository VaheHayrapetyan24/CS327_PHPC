
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void noop() {
    int i = threadIdx.x;
}

__global__ void no_div(int* data, int len) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= len) {
        return;
    }
    data[idx] *= 2;
}

__global__ void diverge(int* data, int len) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= len) {
        return;
    }

    if (idx % 2) {
        data[idx] *= 3;
    } else {
        data[idx] *= 2;
    }
}

__global__ void aligned_div(int* data, int len) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= len) {
        return;
    }

    if ((idx / warpSize) % 2 == 1) {
        data[idx] *= 3;
    } else {
        data[idx] *= 2;
    }
}

void run_kernel(int* ar, int length, char* name, void (*f)(int*, int)) {
    hipEvent_t start, stop;
    float milliseconds = 0;

    dim3 grid, block;
    block = { 256 };
    grid = { (length + block.x - 1) / block.x };


    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    f<<<grid, block>>>(ar, length);
    hipEventRecord(stop);

    hipDeviceSynchronize();

    hipEventElapsedTime(&milliseconds, start, stop);

    printf("%s %f ms\n", name, milliseconds);
}


int main () {
    int length = 100000000;
    int size = sizeof(int) * length;
    int* ar = (int*) malloc(size);

    for (int i = 0; i < length; ++i) {
        ar[i] = i;
    }

    int *ar1, *ar2, *ar3;

    hipMalloc((void **) &ar1, size);
    hipMalloc((void **) &ar2, size);
    hipMalloc((void **) &ar3, size);

    hipMemcpy(ar1, ar, size, hipMemcpyHostToDevice);
    hipMemcpy(ar2, ar1, size, hipMemcpyDeviceToDevice);
    hipMemcpy(ar3, ar1, size, hipMemcpyDeviceToDevice);

    noop<<<10000, 10000>>>();

    run_kernel(ar1, length, "no_div", no_div);
    run_kernel(ar2, length, "diverge", diverge);
    run_kernel(ar3, length, "aligned_div", aligned_div);

    int *ar1_r = (int*) malloc(size);
    int *ar2_r = (int*) malloc(size);
    int *ar3_r = (int*) malloc(size);

    hipMemcpy(ar1_r, ar1, size, hipMemcpyDeviceToHost);
    hipMemcpy(ar2_r, ar2, size, hipMemcpyDeviceToHost);
    hipMemcpy(ar3_r, ar3, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < 50; ++i) {
        printf("%d %d %d\n", ar1_r[i], ar2_r[i], ar3_r[i]);
    }

    hipFree(ar1);
    hipFree(ar2);
    hipFree(ar3);

    free(ar);
    free(ar1_r);
    free(ar2_r);
    free(ar3_r);
}