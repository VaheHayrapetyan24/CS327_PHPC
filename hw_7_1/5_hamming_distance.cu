
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

#define XDIM 1024
#define MUL 4

__global__ void count(char* d_1_in, char* d_2_in, int* d_out, int n) {
    __shared__ int res[XDIM];

    int tid = threadIdx.x;
    char* cur_d_1_in = d_1_in + XDIM * blockIdx.x * MUL;
    char* cur_d_2_in = d_2_in + XDIM * blockIdx.x * MUL;

    int ftid = tid * 4;
    res[tid] = (cur_d_1_in[ftid] != cur_d_2_in[ftid]);
    res[tid] += (cur_d_1_in[ftid + 1] != cur_d_2_in[ftid + 1]);
    res[tid] += (cur_d_1_in[ftid + 2] != cur_d_2_in[ftid + 2]);
    res[tid] += (cur_d_1_in[ftid + 3] != cur_d_2_in[ftid + 3]);
    __syncthreads();

    for (int stride = XDIM / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            res[tid] += res[tid + stride];
        }

        __syncthreads();
    }


    if (tid == 0) {
        d_out[blockIdx.x] = res[0];
    }
}

double cpuSecond() {
    struct timeval tp;
    gettimeofday(&tp,NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}

char get_rand() {
    switch (rand() % 4)
    {
    case 0:
        return 'A';
    case 1:
        return 'C';
    case 2:
        return 'G';
    case 3:
        return 'T';
    }
}

int main() {
    int N = 1 << 29;
    printf("N = %d\n", N);
    size_t in_size = sizeof(char) * N;
    char *h_1_in, *h_2_in;
    char *d_1_in, *d_2_in;
    int *d_res, *h_res, actual_res = 0;

    dim3 grid, block;
    block.x = XDIM;
    grid.x = (N + block.x - 1) / (4 * block.x);
    printf("dimensions<<<%d, %d>>>\n", grid.x, block.x);

    size_t out_size = sizeof(int) * grid.x;
    
    hipError_t err_1_host = hipHostMalloc(&h_1_in, in_size, hipHostMallocDefault);
    hipError_t err_2_host = hipHostMalloc(&h_2_in, in_size, hipHostMallocDefault);
    hipError_t err_3_host = hipHostMalloc(&h_res, out_size, hipHostMallocDefault);
    printf("err host: 1: %s 2: %s 3: %s\n", hipGetErrorString(err_1_host), hipGetErrorString(err_2_host), hipGetErrorString(err_3_host));

    hipError_t err_1_in = hipMalloc(&d_1_in, in_size);
    hipError_t err_2_in = hipMalloc(&d_2_in, in_size);
    hipError_t err_out = hipMalloc(&d_res, out_size);

    printf("err1: %s, err2: %s, out: %s\n", hipGetErrorString(err_1_in), hipGetErrorString(err_2_in), hipGetErrorString(err_out));

    for (int i = 0; i < N; ++i) {
        h_1_in[i] = get_rand();
        h_2_in[i] = get_rand();
    }

    double i_start, i_time;
    
    // STARTS HERE
    i_start = cpuSecond();
    hipMemcpy(d_1_in, h_1_in, in_size, hipMemcpyHostToDevice);
    hipMemcpy(d_2_in, h_2_in, in_size, hipMemcpyHostToDevice);

    count<<<grid, block>>>(d_1_in, d_2_in, d_res, N);

    hipError_t err_sync = hipDeviceSynchronize();
    printf("err sync: %s\n", hipGetErrorString(err_sync));

    hipMemcpy(h_res, d_res, out_size, hipMemcpyDeviceToHost);
    
    int device_sum = 0;
    for (int i = 0; i < grid.x; ++i) {
        device_sum += h_res[i];
    }

    i_time = cpuSecond() - i_start;

    printf("device took %f s\n", i_time);

    // ENDS HERE

    

    i_start = cpuSecond();

    for (int i = 0; i < N; ++i) {
        actual_res += (h_1_in[i] != h_2_in[i]);
    }
    i_time = cpuSecond() - i_start;

    printf("host took %f s\n", i_time);


    printf("\n");

    printf("results: host: %d, device: %d\n",  actual_res, device_sum);

    hipFree(h_1_in);
    hipFree(h_2_in);
    hipFree(d_1_in);
    hipFree(d_2_in);
    hipFree(h_res);
    hipFree(d_res);
}